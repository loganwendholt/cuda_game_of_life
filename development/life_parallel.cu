#include "hip/hip_runtime.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <time.h>
# include <string.h>
# include <hip/hip_runtime.h>
# define GL_GLEXT_PROTOTYPES
/* Include the OpenGL headers */
# include <GL/gl.h>
# include <GL/glext.h>
# include <GL/glu.h>
# include <GL/glut.h>
# include <cuda_gl_interop.h>

/* ----- Function Declarations ----- */
void life_init();
void life_update();
int mod (int a, int b);
void display();
void reshape(int w, int h);
void update();
void graphics_init();
void add_rabbits_pattern(int start_x, int start_y);
struct timespec timer_start();
long timer_end(struct timespec start_time);
void drawTexture();
void displayLifeKernel();

/* ----- Defines ----- */
#define WINDOW_WIDTH 640    // width of window in pixels
#define WINDOW_HEIGHT 480   // height of window in pixels
#define GAME_WIDTH 128      // number of cells it takes to span the window horizontally
#define GAME_HEIGHT 96      // number of cells it takes to span the window vertically

#define NUM_BLOCKS GAME_HEIGHT*GAME_WIDTH/128
#define NUM_THREADS 128

#define WHITE 1.0, 1.0, 1.0 // RGB float values for white, in OpenGL format
#define BLACK 0.0, 0.0, 0.0 // RGB float values for black, in OpenGL format

/* ----- Global Variables ----- */

// Convert the #define parameters to OpenGL-compatible types
GLint window_width = WINDOW_WIDTH;
GLint window_height = WINDOW_HEIGHT;
GLint game_width = GAME_WIDTH;
GLint game_height = GAME_HEIGHT;

// Define the OpenGL vertices for the display area 
GLfloat left = 0.0;
GLfloat right = 1.0;
GLfloat bottom = 0.0;
GLfloat top = 1.0;

// Define the two arrays that contain all cell data
// These arrays will be used in a "ping-pong" fashion,
// where one array will be displayed while another array
// is being filled with updated data 
unsigned char *gridA;
unsigned char *gridB;

// define pointers to allow the grids to be easily swapped
unsigned char *grid;
unsigned char *nextGrid;

// define pointers to GPU memory
unsigned char *lifeData;
unsigned char *nextLifeData;

// global timer used to calculate frame rate
struct timespec frame_timer;

// CUDA stream identifiers
hipStream_t stream1;
hipStream_t stream2;

// Buffer for OpenGL CUDA data
hipGraphicsResource* cudaPboResource;
GLuint gl_pixelBufferObject = 0;
GLuint gl_texturePtr = 0;
unsigned char* d_cpuDisplayData;

// Host-side texture pointer
uchar4* h_textureBufferData;
// Device-side texture pointer.
uchar4* d_textureBufferData;

/* ----- Function Definitions ----- */

/* ***************************************************
*  FUNCTION:  life_kernel
*
*  DESCRIPTION:
*    CUDA kernel to process a single cell on the life grid
*  
*  PARAMETERS:
*    char *sourceGrid:  a reference to the current simulation data
*    char *destGrid:    location of memory to save new simulation data
*
*  RETURN VALUE:
*    none
*
* ****************************************************/
__global__ void life_kernel(unsigned char *sourceGrid, unsigned char *destGrid)
{

  /* Work out our thread id */
  unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
  unsigned int tid = idx + idy * blockDim.x * gridDim.x;

  /* Get the x-y coordinates of the cell being processed */
  unsigned int x = tid % GAME_WIDTH;
  unsigned int y = tid / GAME_WIDTH;

  unsigned int xLeft  = (x-1 + GAME_WIDTH)  % GAME_WIDTH;
  unsigned int xRight = (x+1 + GAME_WIDTH)  % GAME_WIDTH;
  unsigned int yUp    = (y-1 + GAME_HEIGHT) % GAME_HEIGHT;
  unsigned int yDown  = (y+1 + GAME_HEIGHT) % GAME_HEIGHT;

  /* Count the number of live neighbors */
  unsigned int aliveCount = 
      sourceGrid[ yUp   * GAME_WIDTH + xLeft  ] +
      sourceGrid[ y     * GAME_WIDTH + xLeft  ] +
      sourceGrid[ yDown * GAME_WIDTH + xLeft  ] + 
      sourceGrid[ yUp   * GAME_WIDTH + x      ] +
      sourceGrid[ yDown * GAME_WIDTH + x      ] +
      sourceGrid[ yUp   * GAME_WIDTH + xRight ] +
      sourceGrid[ y     * GAME_WIDTH + xRight ] +
      sourceGrid[ yDown * GAME_WIDTH + xRight ]; 

  /* Calculate the next state of the cell */
  destGrid[tid] = aliveCount == 3 || (aliveCount == 2 && sourceGrid[tid]) ? 255 : 0;

}

__global__ void displayLifeKernel(const unsigned char* lifeData, uchar4* destination) 
{
  /* determine thread ID */
  unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
  unsigned int tid = idx + idy * blockDim.x * gridDim.x;

  /* Get cell status */
  unsigned int value = lifeData[tid];

  /* assign values to screen buffer */
  destination[tid].x = value;
  destination[tid].y = value;
  destination[tid].z = value;
  destination[tid].w = value;
}
/* ***************************************************
*  FUNCTION:  gpu_init
*
*  DESCRIPTION:
*    Allocate memory on the GPU 
*  
*  PARAMETERS:
*    none
*     
*  RETURN VALUE:
*    none
*
* ****************************************************/
void gpu_init()
{
  /* Create CUDA stream */
  hipStreamCreate(&stream1);

  /* Allocate memory on the GPU */
  hipMalloc(&lifeData, GAME_WIDTH*GAME_HEIGHT);
  hipMalloc(&nextLifeData, GAME_WIDTH*GAME_HEIGHT);

  /* Transfer data to the GPU */
  hipMemcpy( lifeData,      grid, GAME_WIDTH*GAME_HEIGHT, hipMemcpyHostToDevice );
  hipMemcpy( nextLifeData,  nextGrid, GAME_WIDTH*GAME_HEIGHT, hipMemcpyHostToDevice );
  
  /* Perform initial simulation step */
  life_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(lifeData, nextLifeData); 
  
  /* Swap the GPU arrays */
  unsigned char *tempData;
  tempData = lifeData;
  lifeData = nextLifeData;
  nextLifeData = tempData;
   
  /* transfer results from GPU */
  hipMemcpyAsync(grid, lifeData, GAME_WIDTH*GAME_HEIGHT, hipMemcpyDeviceToHost, stream1 );
 
  /* Perform simulation step */
  life_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream2>>>(lifeData, nextLifeData);
  
  /* start framerate timer */
  frame_timer = timer_start();
}

/* ***************************************************
*  FUNCTION:  runLifeKernel
*
*  DESCRIPTION:
*    Run one step of the simulation through the CUDA kernel
*  
*  PARAMETERS:
*    none
*     
*  RETURN VALUE:
*    none
*
* ****************************************************/
void runLifeKernel()
{
  /* wait for both threads to finish */
  //hipDeviceSynchronize();
 
  /* Swap the GPU arrays */
  unsigned char *tempData;
  tempData = lifeData;
  lifeData = nextLifeData;
  nextLifeData = tempData;
  
  /* run kernel */
  life_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(lifeData, nextLifeData);
  
  /* transfer results from GPU */
  hipMemcpyAsync(nextGrid, lifeData, GAME_WIDTH*GAME_HEIGHT, hipMemcpyDeviceToHost, stream2 );

  /* swap stream references */
  hipStream_t temp;
  temp = stream1;
  stream1 = stream2;
  stream2 = temp;

}

/* ***************************************************
*  FUNCTION:  timer_start
*
*  DESCRIPTION:
*    Begins running a nanosecond-resolution timer
*  
*  PARAMETERS:
*    none
*     
*  RETURN VALUE:
*    struct timespec:  a reference to the timer's start time
*
* ****************************************************/
struct timespec timer_start(){
    struct timespec start_time;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start_time);
    return start_time;
}

/* ***************************************************
*  FUNCTION:  timer_end
*
*  DESCRIPTION:
*    Used to end a nanosecond-resolution timer
*  
*  PARAMETERS:
*    struct timespec start_time: the value returned by the timer_start function
*     
*  RETURN VALUE:
*    long:  The difference in the start and end times, in nanoseconds
*
* ****************************************************/
long timer_end(struct timespec start_time){
    struct timespec end_time;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &end_time);
    long diffInNanos = end_time.tv_nsec - start_time.tv_nsec;
    return diffInNanos;
}
// ----- END TIMING FUNCTIONS ------------------------



/* ***************************************************
*  FUNCTION:  display
*
*  DESCRIPTION:
*    Uses OpenGL to display the current array pointed
*    to by the global variable *grid
*  
*  PARAMETERS:
*    none
*     
*  RETURN VALUE:
*    none
*
* ****************************************************/
void display() {
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glLoadIdentity();
  
  GLfloat xSize = (right - left) / game_width;
  GLfloat ySize = (top - bottom) / game_height;
  
  
  // iterate through the grid and display each cell
  // as either a white or a black square (quad)
  GLint x,y;
  glBegin(GL_QUADS);
  for (x = 0; x < game_width; ++x) 
  {
    for (y = 0; y < game_height; ++y)
    {
      grid[x+y*game_width]?glColor3f(BLACK):glColor3f(WHITE);
            
      glVertex2f(    x*xSize+left,    y*ySize+bottom);
      glVertex2f((x+1)*xSize+left,    y*ySize+bottom);
      glVertex2f((x+1)*xSize+left,(y+1)*ySize+bottom);
      glVertex2f(    x*xSize+left,(y+1)*ySize+bottom);
    }
  }
  glEnd();
      
  glFlush();
  glutSwapBuffers();
 
  /* Swap the host arrays */
  unsigned char *temp;
  temp = grid;
  grid = nextGrid;
  nextGrid = temp;

  /* calculate how long it took to render this frame, then restart counter */ 
  long time_elapsed_nanos = timer_end(frame_timer);
  printf("Frame Time: (nanoseconds): %ld\n", time_elapsed_nanos);

  frame_timer = timer_start();
  
}

void cudaDisplay()
{
  printf("Entering display...\n");
  hipGraphicsMapResources(1, &cudaPboResource, 0);
 
  size_t num_bytes;
  hipGraphicsResourceGetMappedPointer((void**)&d_textureBufferData, &num_bytes, cudaPboResource);

  displayLifeKernel<<<NUM_BLOCKS, NUM_THREADS>>>(grid, d_textureBufferData);
  hipDeviceSynchronize();

  hipGraphicsUnmapResources(1, &cudaPboResource, 0);
  
  drawTexture();
}

void drawTexture()
{
  glColor3f(WHITE);
  glBindTexture(GL_TEXTURE_2D, gl_texturePtr);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject);

  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, window_width, window_height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

  glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f);
    glVertex2f(0.0f, 0.0f);
    glTexCoord2f(1.0f, 0.0f);
    glVertex2f(float(window_width), 0.0f);
    glTexCoord2f(1.0f, 1.0f);
    glVertex2f(float(window_width), float(window_height));
    glTexCoord2f(0.0f, 1.0f);
    glVertex2f(0.0f, float(window_height));
  glEnd();

  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
  glBindTexture(GL_TEXTURE_2D, 0);
}

bool initOpenGlBuffers(int width, int height) 
{

printf("Check1\n");
  glDeleteTextures(1, &gl_texturePtr);
  gl_texturePtr = 0;

printf("Check2\n");
  if (gl_pixelBufferObject) 
  {
    hipGraphicsUnregisterResource(cudaPboResource);
    glDeleteBuffers(1, &gl_pixelBufferObject);
    gl_pixelBufferObject = 0;
  }

printf("Check3\n");
  
  glEnable(GL_TEXTURE_2D);
  
  glGenTextures(1, &gl_texturePtr);
  glBindTexture(GL_TEXTURE_2D, gl_texturePtr);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, h_textureBufferData);
printf("3b\n");
  glGenBuffers(1, &gl_pixelBufferObject);
printf("3c\n");
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pixelBufferObject);
printf("3d\n");
  glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width * height * sizeof(uchar4), h_textureBufferData, GL_STREAM_COPY);

printf("Check4\n");		
  hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, gl_pixelBufferObject, cudaGraphicsMapFlagsWriteDiscard);
  if (result != hipSuccess) {
    return false;
  }

  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
  glBindTexture(GL_TEXTURE_2D, 0);
  return true;
}


/* ***************************************************
*  FUNCTION:  reshape
*
*  DESCRIPTION:
*    Callback function used to handle any reshaping of the 
*    display window
*  
*  PARAMETERS:
*    int w:    new width
*    int h:    new height
*     
*  RETURN VALUE:
*    none
*
* ****************************************************/
void reshape(int w, int h) {
  printf("Entering reshape\n");
  window_width = w;
  window_height = h;

  glViewport(0, 0, window_width, window_height);

  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluOrtho2D(left, right, bottom, top);

  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
  printf("initializing buffers\n");

  initOpenGlBuffers(window_width, window_height);

  glutPostRedisplay();
  printf("Exiting reshape\n");
}

/* ***************************************************
*  FUNCTION:  update
*
*  DESCRIPTION:
*    Callback function that calls the Game of Life 
*    siimulation function, then updates the screen
*    with the results.
*  
*  PARAMETERS:
*    none
*     
*  RETURN VALUE:
*    none
*
* ****************************************************/
void update() {
  struct timespec vartime = timer_start();
  runLifeKernel();
  long time_elapsed_nanos = timer_end(vartime);
  printf("Time taken (nanoseconds): %ld\n", time_elapsed_nanos);
  glutPostRedisplay();
}

/* ***************************************************
*  FUNCTION:  graphics_init
*
*  DESCRIPTION:
*    Handles window initialization and binding of
*    OpenGL/glut callback functions
*  
*  PARAMETERS:
*    none
*     
*  RETURN VALUE:
*    none
*
* ****************************************************/
void graphics_init()
{
  
  glutInitWindowSize(window_width, window_height);
  glutInitWindowPosition(0, 0);
  glutCreateWindow("Game of Life");
  glClearColor(1, 1, 1, 1);
  
  glutReshapeFunc(reshape);
  glutDisplayFunc(cudaDisplay);
  glutIdleFunc(update);
}

/* ***************************************************
*  FUNCTION:  add_rabbits_pattern
*
*  DESCRIPTION:
*    Adds the "rabbits" pattern to the current grid
*  
*  PARAMETERS:
*    int start_x:  the x origin of the pattern (starting from left)
*    int start_y:  the y origin of the pattern (starting from top)
*
*  RETURN VALUE:
*    none
*
* ****************************************************/
void add_rabbits_pattern(int start_x, int start_y)
{
  int rabbits_pattern[18] = {0,0, 4,0, 5,0, 6,0, 0,1, 1,1, 2,1, 5,1, 1,2};
  int x, y, i;
  
  for(i=0; i<18; i+=2)
  {
    x = (rabbits_pattern[i] + start_x + game_width) % game_width;
    y = (rabbits_pattern[i+1] + start_y + game_height) % game_height;

    grid[y*game_width+x] = 255;
  }
}


/* ***************************************************
*  FUNCTION:  life_init
*
*  DESCRIPTION:
*    Initializes the arrays used to keep track of 
*    cell data. Zeros out both "cell buffers", then
*    applies a pattern to generate some life
*  
*  PARAMETERS:
*    none
*
*  RETURN VALUE:
*    none
*
* ****************************************************/
void life_init()
{
  int i, j;
printf("before\n");
  /* Allocate page-locked memory on the host */
  hipError_t status = hipHostMalloc((void**)&gridA, GAME_WIDTH*GAME_HEIGHT);
  if (status != hipSuccess)
    printf("Error allocating pinned host memory\n");  
  
  status = hipHostMalloc((void**)&gridB, GAME_WIDTH*GAME_HEIGHT);
  if (status != hipSuccess)
    printf("Error allocating pinned host memory\n");  
  printf("after\n");

  grid = gridA;
  nextGrid = gridB;

  // zero out both buffers
  for( i = 0; i < game_height; i++ )
  {
    for( j = 0; j < game_width; j++ )
    {
      printf("%d %d \n", i,j);
      gridA[i*game_width+j] = 0;
      gridB[i*game_width+j] = 0;
    }
  }
  printf("adding rabbits\n");
  // add a pattern to the buffer
  add_rabbits_pattern(game_width/2-3,game_height/2-3);
printf("done\n");
}

/* ***************************************************
*  FUNCTION:  main
*
*  DESCRIPTION:
*    Main program execution loop. All actual looping
*    is handled by glut, so all calculations that
*    need to occur within the loop must be set within
*    glut callbacks. 
*  
*  PARAMETERS:
*    int argc:     Number of input parameters (unused)
*    char **argv:  Input parameters (unused)
*
*  RETURN VALUE:
*    int:  return status
*
* ****************************************************/
int main(int argc, char **argv)
{
  // Initialize OpenGL/GLUT
  glutInit(&argc, argv);
  graphics_init();

  // Initialize array
  life_init();

  // Initialize GPU
  gpu_init();
  printf("GPU initialized\n");
  // Begin main loop
  glutMainLoop();

  return 0;
}
  
